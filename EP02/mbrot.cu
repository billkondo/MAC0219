#include "hip/hip_runtime.h"
#include <thrust/complex.h>
#include <stdio.h> 
#include <stdlib.h>
#include <png.h>
#include <sys/time.h>

#define M 200

#define DIE(...) { \
  fprintf(stderr, __VA_ARGS__); \
  exit(EXIT_FAILURE); \
}

float points[4];  // c0_real, c0_image, c1_real, c1_image
int w, h;
char cpu_gpu[5];  // do the calculations in cpu or gpu  
int num_threads_arg;
char saida[256];

void application_error(char message[256]) {
  printf ("%s\n", message);
  exit(0);
}

void write_png_file(float *buffer) {
  int i, j;
  png_bytep row = NULL;
  png_structp png_ptr;
  png_infop info_ptr;

  /* Open file for writing (binary mode) */
  FILE *fp = fopen(saida, "wb");
  if (!fp) {
    DIE("write_png_file(): Could not open file ... ");
  }

  /* Initialize write structure */
  png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  if (!png_ptr) {
    DIE("write_png_file(): png_ptr is NULL ... ");
  }
  
  /* Initialize info structure */
  info_ptr = png_create_info_struct(png_ptr);
  if (!info_ptr) {
    if (png_ptr != NULL) png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
    DIE("write_png_file(): info_ptr is NULL ... ");
  }

  /* Exception handling */
  if (setjmp(png_jmpbuf(png_ptr))) {
    if (info_ptr != NULL) png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
    if (png_ptr != NULL) png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
    if (fp != NULL) fclose(fp);
    if (row != NULL) free(row);
    DIE("write_png_file(): libpng error ... ");
  }

  png_init_io(png_ptr, fp);

  png_set_IHDR(png_ptr, info_ptr, w, h,
			8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
			PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

  png_write_info(png_ptr, info_ptr);

  row = (png_bytep) malloc(3 * w * sizeof(png_byte));

  /* write image data */
  for (i = 0; i < h; ++i) {
    for (j = 0; j < w; ++j) {
      row[3 * j] = buffer[i * w + j];
      row[3 * j + 1] = 0;
      row[3 * j + 2] =  0;
    }
    png_write_row(png_ptr, row);
  }

  /* End write */
  png_write_end(png_ptr, NULL);

  if (fp != NULL) fclose(fp);
	if (info_ptr != NULL) png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
	if (png_ptr != NULL) png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
	if (row != NULL) free(row); 
}

float test_complex_number(int k) {
  int j = k / w;
  int i = k % w;

  float dx = (points[2] - points[0]) / w ;
  float dy = (points[3] - points[1]) / h;
  float image = points[1] + dy * i;
  float real = points[0] + dx * j;

  thrust::complex<float> c = thrust::complex<float>(real, image);
  thrust::complex<float> z = thrust::complex<float>(0, 0);
  int it = 0;

  while (it < M && norm(z) < 2.0) {
    z = z * z + c;
    ++it;
  }

  if (it < M) {
    return (255 * it) / M;
  } 
  
  return 0;
}

float *cpu_create_image() {
  float *buffer = (float *) malloc(sizeof(float) * w * h);
  int k;

  if (buffer == NULL) {
    DIE("createImage(): could create buffer ... ");
  }

  #pragma omp parallel for num_threads(num_threads_arg) 
  for (k = 0; k < w * h; ++k)
    buffer[k] = test_complex_number(k);
 
  return buffer;
}

void cpu_solve() {
  float *buffer;
  buffer = cpu_create_image();
  write_png_file(buffer);
  free(buffer);
}

__global__
void gpu_create_image(float *buffer, int w, int h, float *points) {
  const int globalIndex = blockDim.x*blockIdx.x + threadIdx.x;
  
  if (globalIndex < w * h) {
    int j = globalIndex / w;
    int i = globalIndex % w;

    float dx = (points[2] - points[0]) / w ;
    float dy = (points[3] - points[1]) / h;
    float image = points[1] + dy * i;
    float real = points[0] + dx * j;

    thrust::complex<float> c = thrust::complex<float>(real, image);
    thrust::complex<float> z = thrust::complex<float>(0, 0);

    int it = 0;

    while (it < M && norm(z) < 2.0) {
      z = z * z + c;
      ++it;
    }

    if (it < M) buffer[globalIndex] = (255 * it) / M;
    else buffer[globalIndex] = 0;
  } 
}


void gpu_solve() {
  const int THREADS_PER_BLOCK = num_threads_arg;
  const int NUM_BLOCKS = (w * h + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  float *buffer_cpu;
  float *buffer_gpu, *points_gpu;
 
  buffer_cpu = (float *) malloc(sizeof(float) * w * h);

  if (buffer_cpu == NULL) {
    DIE("gpu_solve(): buffer_cpu is NULL ...");
  }

  hipMalloc(&buffer_gpu, w * h * sizeof(float));
  hipMalloc(&points_gpu, 4 * sizeof(float));

  hipMemcpy(points_gpu, points, 4 * sizeof(float), hipMemcpyHostToDevice);

  gpu_create_image<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(buffer_gpu, w, h, points_gpu);
  hipDeviceSynchronize();

  hipMemcpy(buffer_cpu, buffer_gpu, w * h * sizeof(float), hipMemcpyDeviceToHost);

  write_png_file(buffer_cpu);

  hipFree(buffer_gpu);
  hipFree(points);
}

int main(int argc, char **argv) {

  struct timeval start, end;

  // Read Arguments
  if (argc < 9 
  || sscanf(argv[1], "%f", &points[0])        != 1 
  || sscanf(argv[2], "%f", &points[1])        != 1
  || sscanf(argv[3], "%f", &points[2])        != 1 
  || sscanf(argv[4], "%f", &points[3])        != 1 
  || sscanf(argv[5], "%d", &w)                != 1 
  || sscanf(argv[6], "%d", &h)                != 1
  || sscanf(argv[7], "%s", cpu_gpu)           != 1 
  || sscanf(argv[8], "%d", &num_threads_arg)  != 1
  || sscanf(argv[9], "%s", saida)             != 1) {
    DIE("Invalid Arguments ...\n");
  }

  gettimeofday(&start, NULL);
  if (cpu_gpu[0] == 'c') cpu_solve();
  else gpu_solve();
  gettimeofday(&end, NULL);

  double elapsed_time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 10000000.0;
  printf("%.4lf\n", elapsed_time);

  return 0;
}